#include "hip/hip_runtime.h"
#include ""
//#include "hip/device_functions.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <time.h>
#include "Header.cuh"
#include "hip/hip_runtime.h"

__global__ void divOnSqrtFFT(hipDoubleComplex *inputSignal, double *sqrtFFT_SIZE)
{
	int index = threadIdx.x + blockIdx.x*blockDim.x;
	//printf("CUDA [%d]: %.5f \n", index, sqrtFFT_SIZE[0]);
	if(index<FFT_size)
	{
		inputSignal[index] = hipCdiv(inputSignal[index], make_hipDoubleComplex(sqrtFFT_SIZE[0], 0));
	}
}

__global__ void mulOnSqrtFFT(hipDoubleComplex *inputSignal, double *sqrtFFT_SIZE)
{
	int index = threadIdx.x + blockIdx.x*blockDim.x;
	if(index<FFT_size)
	{
		inputSignal[index] = hipCmul(inputSignal[index], make_hipDoubleComplex(sqrtFFT_SIZE[0],0));
	}
}

__global__ void deviceDecodeQAM(hipDoubleComplex *signal)
{
	int index = threadIdx.x + blockIdx.x*blockDim.x;
	if(index<FFT_size)
	{
		double real = hipCreal(signal[index]);
		double imag = hipCimag(signal[index]);
		//printf("index: %d REAL %.2f, IMAG %.2f\n", index, real, imag);
		if (real>0)
			real = 1;
		else
			real = -1;
		if (imag>0)
			imag = 1;
		else
			imag = -1;
			signal[index] = make_hipDoubleComplex(real, imag);
	}
}

__global__ void deviceModulate(hipDoubleComplex *signal)
{
	int index = threadIdx.x + blockIdx.x*blockDim.x;
	if(index<FFT_size)
	{
		double real = hipCreal(signal[index]);
		double imag = hipCimag(signal[index]);

		if (real > 0.5)
			real = 1;
		else
			real = -1;
		if (imag > 0.5)
			imag = 1;
		else
			imag = -1;
		signal[index] = make_hipDoubleComplex(real, imag);
	}
}

__global__ void subtract_SIC(hipDoubleComplex *timeSignal, hipDoubleComplex *signalArray, double *coefficientsArray)
{
	int index = threadIdx.x + blockIdx.x*blockDim.x;
	if(index<FFT_size)
	{
		timeSignal[index]  = hipCmul(timeSignal[index], make_hipDoubleComplex(sqrt((double)coefficientsArray[0]), 0));
		timeSignal[index]  = hipCmul(timeSignal[index], make_hipDoubleComplex((double)-1, 0));
		signalArray[index] = hipCadd(signalArray[index], timeSignal[index]);
	}
}

SignalAndSpendTime deviceDecodeQAM_wrapper (hipDoubleComplex *rxCarrierSignal)
{
	hipEvent_t start, stop;
	float elapsedTime;
	hipDoubleComplex *dev_rxCarrierSignal;
	hipMalloc((void**)&dev_rxCarrierSignal, FFT_size*sizeof(hipDoubleComplex));
	hipMemcpy(dev_rxCarrierSignal, rxCarrierSignal, FFT_size*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	deviceDecodeQAM << < FFT_size, UEs >> >(dev_rxCarrierSignal);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	hipDoubleComplex *inputSignalResult;
	inputSignalResult = (hipDoubleComplex *)malloc(FFT_size*sizeof(hipDoubleComplex));
	hipMemcpy(inputSignalResult, dev_rxCarrierSignal, FFT_size*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
	hipFree(dev_rxCarrierSignal);
	SignalAndSpendTime result = { inputSignalResult, elapsedTime };
	return result;
}

SignalAndSpendTime deviceModulate_wrapper (hipDoubleComplex *rxCarrierSignal)
{
	hipEvent_t start, stop;
	float elapsedTime;
	hipDoubleComplex *dev_rxCarrierSignal;
	hipMalloc((void**)&dev_rxCarrierSignal, FFT_size*sizeof(hipDoubleComplex));
	hipMemcpy(dev_rxCarrierSignal, rxCarrierSignal, FFT_size*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	deviceModulate << < 8, FFT_size, UEs >> >(dev_rxCarrierSignal);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	hipDoubleComplex *inputSignalResult;
	inputSignalResult = (hipDoubleComplex *)malloc(FFT_size*sizeof(hipDoubleComplex));
	hipMemcpy(inputSignalResult, dev_rxCarrierSignal, FFT_size*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
	hipFree(dev_rxCarrierSignal);
	SignalAndSpendTime result = { inputSignalResult, elapsedTime };
	return result;
}

SignalAndSpendTime divOnSqrtFFT_wrapper (hipDoubleComplex *inputSignal)
{
	hipEvent_t start, stop;
	float elapsedTime;
	double *sqrtFFT_SIZE;
	sqrtFFT_SIZE = (double*)malloc(sizeof(double));
	sqrtFFT_SIZE[0] = sqrt(FFT_size);
	double *dev_sqrtFFT_SIZE;
	hipMalloc((void**)&dev_sqrtFFT_SIZE, sizeof(double));
	hipMemcpy(dev_sqrtFFT_SIZE, sqrtFFT_SIZE, sizeof(double), hipMemcpyHostToDevice);
	hipDoubleComplex *dev_inputSignal;
	hipMalloc((void**)&dev_inputSignal, FFT_size*sizeof(hipDoubleComplex));
	hipMemcpy(dev_inputSignal, inputSignal, FFT_size*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	divOnSqrtFFT << < FFT_size, UEs >> >(dev_inputSignal, dev_sqrtFFT_SIZE);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	hipDoubleComplex *inputSignalResult;
	inputSignalResult = (hipDoubleComplex *)malloc(FFT_size*sizeof(hipDoubleComplex));
	hipMemcpy(inputSignalResult, dev_inputSignal, FFT_size*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
	hipFree(dev_inputSignal);
	hipFree(dev_sqrtFFT_SIZE);
	SignalAndSpendTime result = { inputSignalResult, elapsedTime };
	return result;
}

SignalAndSpendTime mulOnSqrtFFT_wrapper (hipDoubleComplex *inputSignal)
{
	hipEvent_t start, stop;
	float elapsedTime;
	double *sqrtFFT_SIZE;
	sqrtFFT_SIZE = (double*)malloc(sizeof(double));
	sqrtFFT_SIZE[0] = sqrt(FFT_size);
	double *dev_sqrtFFT_SIZE;
	hipMalloc((void**)&dev_sqrtFFT_SIZE, sizeof(double));
	hipMemcpy(dev_sqrtFFT_SIZE, sqrtFFT_SIZE, sizeof(double), hipMemcpyHostToDevice);
	hipDoubleComplex *dev_inputSignal;
	hipMalloc((void**)&dev_inputSignal, FFT_size*sizeof(hipDoubleComplex));
	hipMemcpy(dev_inputSignal, inputSignal, FFT_size*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	mulOnSqrtFFT << < FFT_size, UEs >> >(dev_inputSignal, dev_sqrtFFT_SIZE);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	hipDoubleComplex *inputSignalResult;
	inputSignalResult = (hipDoubleComplex *)malloc(FFT_size*sizeof(hipDoubleComplex));
	hipMemcpy(inputSignalResult, dev_inputSignal, FFT_size*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
	hipFree(dev_inputSignal);
	hipFree(dev_sqrtFFT_SIZE);
	SignalAndSpendTime result = { inputSignalResult, elapsedTime };
	return result;
}

__global__ void subtract(hipDoubleComplex *signalArray, hipDoubleComplex *sumTimeSignal)
{
	int index = threadIdx.x + blockIdx.x*blockDim.x;
	if(index<FFT_size)
	{
		signalArray[index] = hipCadd(signalArray[index], hipCmul(sumTimeSignal[index], make_hipDoubleComplex((double)-1, 0)));
	}
}

SignalAndSpendTime subtract_wrapper (hipDoubleComplex *inputSignal, hipDoubleComplex *sumTimeSignal)
{
	hipEvent_t start, stop;
	float elapsedTime;

	hipDoubleComplex *dev_inputSignal;
	hipMalloc((void**)&dev_inputSignal, FFT_size*sizeof(hipDoubleComplex));
	hipMemcpy(dev_inputSignal, inputSignal, FFT_size*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);

	hipDoubleComplex *dev_sumSignal;
	hipMalloc((void**)&dev_sumSignal, FFT_size*sizeof(hipDoubleComplex));
	hipMemcpy(dev_sumSignal, sumTimeSignal, FFT_size*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);

	//printf("test message from subtract_wrapper\n");
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	subtract << < FFT_size, UEs  >> >(dev_inputSignal, dev_sumSignal);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	hipDoubleComplex *inputSignalResult;
	inputSignalResult = (hipDoubleComplex *)malloc(FFT_size*sizeof(hipDoubleComplex));
	hipMemcpy(inputSignalResult, dev_inputSignal, FFT_size*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
	hipFree(dev_inputSignal);
	hipFree(dev_sumSignal);

	SignalAndSpendTime result = { inputSignalResult, elapsedTime };
	return result;
}

__global__ void getTimeSignal(hipDoubleComplex *dev_timeSignalArray, hipDoubleComplex *rxCarrier, double *sqrtCoefficient)
{
	int b_index = blockIdx.x;
	if(b_index<FFT_size && threadIdx.x == 0)
	{
		dev_timeSignalArray[b_index] = hipCmul(rxCarrier[b_index], make_hipDoubleComplex((double)sqrtCoefficient[0], 0));
	}
	__syncthreads();
	if(b_index<FFT_size && threadIdx.x == 0)
	{
		dev_timeSignalArray[b_index] = hipCmul(dev_timeSignalArray[b_index], make_hipDoubleComplex(UEs, 0));
	}
}

SignalAndSpendTime sum_wrapper (hipDoubleComplex *rxCarrierSignal)
{
	hipEvent_t start, stop;
	float elapsedTime;

	double *sqrtCoefficient;
	sqrtCoefficient = (double*)malloc(sizeof(double));
	sqrtCoefficient[0] = sqrt((double)coefficientsArray[0]);

	double *dev_sqrtCoefficient;
	hipMalloc((void**)&dev_sqrtCoefficient, sizeof(double));
	hipMemcpy(dev_sqrtCoefficient, sqrtCoefficient, sizeof(double), hipMemcpyHostToDevice);

	hipDoubleComplex *dev_rxCarrierSignal;
	hipMalloc((void**)&dev_rxCarrierSignal, FFT_size*sizeof(hipDoubleComplex));
	hipMemcpy(dev_rxCarrierSignal, rxCarrierSignal, FFT_size*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);

	hipDoubleComplex *dev_timeSignalArray;
	hipMalloc((void**)&dev_timeSignalArray, FFT_size*sizeof(hipDoubleComplex));

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	getTimeSignal << < FFT_size, 1  >> >(dev_timeSignalArray, dev_rxCarrierSignal, dev_sqrtCoefficient);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);

	hipDoubleComplex *inputSignalResult;
	inputSignalResult = (hipDoubleComplex *)malloc(FFT_size*sizeof(hipDoubleComplex));
	hipMemcpy(inputSignalResult, dev_timeSignalArray, FFT_size*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);

	hipFree(dev_rxCarrierSignal);
	hipFree(dev_sqrtCoefficient);
	hipFree(dev_timeSignalArray);

	SignalAndSpendTime result = { inputSignalResult, elapsedTime };
	return result;
}

SignalAndSpendTime subtract_SIC_wrapper (hipDoubleComplex *timeSignal, hipDoubleComplex *inputSignal, int a)
{
	hipEvent_t start, stop;
	float elapsedTime;

	hipDoubleComplex *dev_inputSignal;
	hipMalloc((void**)&dev_inputSignal, FFT_size*sizeof(hipDoubleComplex));
	hipMemcpy(dev_inputSignal, inputSignal, FFT_size*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);

	hipDoubleComplex *dev_timeSignal;
	hipMalloc((void**)&dev_timeSignal, FFT_size*sizeof(hipDoubleComplex));
	hipMemcpy(dev_timeSignal, timeSignal, FFT_size*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);

	double *sqrtCoefficient;
	sqrtCoefficient = (double*)malloc(sizeof(double));
	sqrtCoefficient[0] = sqrt((double)coefficientsArray[UEs-a-1]);

	double *dev_sqrtCoefficient;
	hipMalloc((void**)&dev_sqrtCoefficient, sizeof(double));
	hipMemcpy(dev_sqrtCoefficient, sqrtCoefficient, sizeof(double), hipMemcpyHostToDevice);

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	subtract_SIC << < FFT_size, UEs >> >(dev_timeSignal, dev_inputSignal, dev_sqrtCoefficient);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);

	hipDoubleComplex *inputSignalResult;
	inputSignalResult = (hipDoubleComplex *)malloc(FFT_size*sizeof(hipDoubleComplex));
	hipMemcpy(inputSignalResult, dev_inputSignal, FFT_size*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);

	hipFree(dev_inputSignal);
	hipFree(dev_timeSignal);
	hipFree(dev_sqrtCoefficient);

	SignalAndSpendTime result = { inputSignalResult, elapsedTime };
	return result;
}

int main()
{
	// I. prepare NOMA signal
	srand((unsigned)time(0));
	coefficientsFill();
	hipDoubleComplex * iFFTsignal;
	iFFTsignal = (hipDoubleComplex *)malloc(FFT_size*sizeof(hipDoubleComplex));

	hipDoubleComplex * signal;
	signal = (hipDoubleComplex *)malloc(FFT_size*sizeof(hipDoubleComplex));

	double sqrtFFT = sqrt((double)FFT_size);
	double sqrtCoeff;
	sqrtCoeff = sqrt((double)coefficientsArray[1]);
	for (int b = 0; b < FFT_size; b++)
	{
		signal[b] = getModulatedSignal();
	}
	iFFTsignal = getiFFT_Main(signal);
	for (int c = 0; c < FFT_size; c++)
	{
		signal[c] = hipCmul(iFFTsignal[c], make_hipDoubleComplex(sqrtCoeff*sqrtFFT, 0));
	}
	// II. send this signal to PIC and/or SIC functions
	double PIC_time = PIC(signal);
	printf("PIC time: %.5f ms\n", PIC_time);
	//double SIC_time = SIC(signal);
	//printf("SIC time: %.5f ms\n", SIC_time);
	return 0;
}
