#include "hip/hip_runtime.h"
#include <hip/hip_complex.h>
#ifndef __HIPCC__
	#define __HIPCC__
#endif
#include "hip/hip_runtime.h"
#include ""
#include <hip/device_functions.h>
#include <stdio.h>
#include <ctime>
#include <time.h>
#include <cstdlib>
#include "Header.cuh"

__device__ hipDoubleComplex decodeQAM4(hipDoubleComplex signal)
{
	float real = hipCreal(signal);
	float imag = hipCimag(signal);

	//printf("REAL %.2f, IMAG %.2f\n", real, imag);

	if (real>0)
		real = 1;
	else
		real = -1;
	if (imag>0)
		imag = 1;
	else
		imag = -1;
	signal = make_hipDoubleComplex(real, imag);
	return signal;
}

__device__ hipDoubleComplex decodeQAM16(hipDoubleComplex signal)
{
	float real = hipCreal(signal);
	float imag = hipCimag(signal);

	//printf("REAL %.2f, IMAG %.2f\n", real, imag);

	if (real >= 0  && real < 2)
		real = 1;
	else if (real >= 2)
		real = 3;
	else if (real < 0  && real > -2)
		real = -1;
	else if (real <= -2)
		real = -3;

if (imag >= 0 && imag < 2)
	imag = 1;
else if (imag >= 2)
imag = 3;
else if (imag < 0 && imag > -2)
imag = -1;
else if (imag <= -2)
imag = -3;

signal = make_hipDoubleComplex(real, imag);
return signal;
}

__global__ void PIC(hipDoubleComplex * channel, hipDoubleComplex * signal)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int index = threadIdx.x;
	__shared__ hipDoubleComplex decodedSignal[UEsPerCluster];
	__shared__ hipDoubleComplex channelWithPower[UEsPerCluster];
	__shared__ hipDoubleComplex cache1[UEsPerCluster];
	__shared__ hipDoubleComplex cache2[UEsPerCluster];
	//__shared__ hipDoubleComplex decodedMessage[UEsPerCluster];
	if (i < numberOfUEs)
	{
		//printf("undecoded signal: %d %.2f, %.2f\n", index, hipCreal(signal[index]), hipCimag(signal[index]));
		float power = (float)1 / (float)numberOfUEs;
		channelWithPower[index] = hipCmul(channel[i], make_hipDoubleComplex(power, 0));
		//printf("%d channelWithPower %.2f, %.2f\n", index, hipCreal(channelWithPower[index]), hipCimag(channelWithPower[index]));
		if (modulation == 4)
			decodedSignal[index] = decodeQAM4(hipCdiv(*signal, channelWithPower[index]));
		if (modulation == 16)
			decodedSignal[index] = decodeQAM16(hipCdiv(*signal, channelWithPower[index]));
		//printf("%d decodedSignal %.2f, %.2f\n", index, hipCreal(decodedSignal[index]), hipCimag(decodedSignal[index]));
	}
	__syncthreads();
	hipDoubleComplex temp1 = make_hipDoubleComplex(0, 0);
	hipDoubleComplex temp2 = make_hipDoubleComplex(0, 0);
	while (index < UEsPerCluster)
	{
		temp1 = hipCadd(temp1, decodedSignal[index]);
		temp2 = hipCadd(temp2, channelWithPower[index]);
		index++;
	}
	index = threadIdx.x;
	cache1[index] = temp1;
	cache2[index] = temp2;
	/*
	__syncthreads();
	if (index == 0)
	{
	printf("sumDecodedSignal %.2f, %.2f\n", hipCreal(cache1[0]), hipCimag(cache1[0]));
	printf("sumChannelWithPower %.2f, %.2f\n", hipCreal(cache2[0]), hipCimag(cache2[0]));
	}
	*/
	__syncthreads();

	if (i < numberOfUEs)
	{
		if (modulation == 4)
			decodeQAM4(hipCadd((hipCdiv(*signal, cache2[0]), cache1[0]), decodedSignal[index]));
		//decodedMessage[i] = decodeQAM4(hipCadd((hipCdiv(*signal, cache2[0]), cache1[0]), decodedSignal[i]));
		if (modulation == 16)
			decodeQAM16(hipCadd((hipCdiv(*signal, cache2[0]), cache1[0]), decodedSignal[index]));
		//decodedMessage[i] = decodeQAM16(hipCadd((hipCdiv(*signal, cache2[0]), cache1[0]), decodedSignal[i]));
		//printf("%d decodedMessage %.2f, %.2f\n", i, hipCreal(decodedMessage[i]), hipCimag(decodedMessage[i]));
	}
}

__global__ void SIC(hipDoubleComplex * channel, hipDoubleComplex * signal)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int index = threadIdx.x;
	__shared__ hipDoubleComplex decodedSignal[UEsPerCluster];
	__shared__ hipDoubleComplex channelWithPower[UEsPerCluster];

	if (i < numberOfUEs)
	{
		//printf("undecoded signal: %d %.2f, %.2f\n", index, hipCreal(signal[index]), hipCimag(signal[index]));
		float power = (float)1 / (float)numberOfUEs;
		for (int i = UEsPerCluster; i >= 0; i--)
		{
			decodedSignal[index] = hipCdiv(hipCsub(*signal, channelWithPower[index]), hipCmul(channel[i], make_hipDoubleComplex(power, 0)));
			if (modulation == 4)
				decodedSignal[index] = decodeQAM4(decodedSignal[index]);
			if (modulation == 16)
				decodedSignal[index] = decodeQAM16(decodedSignal[index]);
			if (i != UEsPerCluster)
			{
				channelWithPower[index] = hipCadd(channelWithPower[index], hipCmul(channel[i], hipCmul(decodedSignal[index], make_hipDoubleComplex(power, 0))));
			}
		}
		//printf("decoded SIC signal: %d %.2f, %.2f\n", index, hipCreal(decodedSignal[index]), hipCimag(decodedSignal[index]));
	}
}

int main()
{
	hipDoubleComplex * channel = getChannel();
	hipDoubleComplex signal = getSignal(channel);
	hipDoubleComplex signalArray[numberOfUEs];
	for (int a = 0; a < numberOfUEs; a++)
	{
		if (ic == 'S')
			signalArray[a] = getSignal(channel);
		if (ic == 'P')
			signalArray[a] = signal;
		//printf("generated signal: %d %.2f, %.2f\n", a, hipCreal(signalArray[a]), hipCimag(signalArray[a]));
	}

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	hipDoubleComplex *dev_channel;
	hipDoubleComplex *dev_signal;

	hipMalloc((void**)&dev_channel, numberOfUEs*sizeof(hipDoubleComplex));
	hipMalloc((void**)&dev_signal,  numberOfUEs*sizeof(hipDoubleComplex));

	hipMemcpy(dev_channel, channel, numberOfUEs*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
	hipMemcpy(dev_signal,  signalArray, numberOfUEs*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
	
	if (ic == 'P')
		PIC << < clusters, UEsPerCluster >> >(dev_channel, dev_signal);
	if (ic == 'S')
		SIC << < clusters, UEsPerCluster >> >(dev_channel, dev_signal);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("cancellation: %cIC; modulation: %d; UEs per cluster: %d; all UEs: %d; elapsed time: %.5f ms; \n", ic, modulation, UEsPerCluster, numberOfUEs, elapsedTime);
	hipFree(dev_channel);
	hipFree(dev_signal);

	return 0;
}
